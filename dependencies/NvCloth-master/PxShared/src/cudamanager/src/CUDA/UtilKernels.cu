#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 2008-2017 NVIDIA Corporation. All rights reserved.

#include "cudamanager/PxGpuCopyDesc.h"
#include "foundation/PxSimpleTypes.h"

#include <hip/hip_runtime.h>

using namespace physx;

extern "C" __host__ void initUtilKernels() {}

extern "C" __global__
void Saturate( )
{
    // NOP
}

__device__
void performCopy( const physx::PxGpuCopyDesc& desc, uint32_t totalBlocks )
{
	if( desc.type == physx::PxGpuCopyDesc::DeviceMemset32 )
	{
		uint32_t *dest = (uint32_t*) desc.dest;
		uint32_t wordCount = desc.bytes >> 2;
		size_t word = blockIdx.x * blockDim.x + threadIdx.x;
		size_t stride = blockDim.x * totalBlocks;
		for( ; word < wordCount ; word += stride )
			dest[ word ] = desc.source;
		return;
	}

	/* The idea here is to maximize throughput with minimal register and thread counts */
	/* Manually unrolled 4 times, the compiler refuses to do it for me */

	if( (desc.source & 0x7) != 0 || (desc.dest & 0x7) != 0 || (desc.bytes & 0x7) != 0)
	{
		/* Input is word aligned */

		uint32_t *dest = (uint32_t*) desc.dest;
		uint32_t *source = (uint32_t*) desc.source;
		uint32_t wordCount = desc.bytes >> 2;
		size_t word = blockIdx.x * blockDim.x + threadIdx.x;
		size_t stride = blockDim.x * totalBlocks;
		while( word < wordCount )
		{
			uint32_t a0, a1, a2, a3, a4, a5;
			a0 = source[ word ];
			if( word + stride < wordCount )
				a1 = source[ word + stride ];
			if( word + stride*2 < wordCount )
				a2 = source[ word + stride*2 ];
			if( word + stride*3 < wordCount )
				a3 = source[ word + stride*3 ];
			if( word + stride*4 < wordCount )
				a4 = source[ word + stride*4 ];
			if( word + stride*5 < wordCount )
				a5 = source[ word + stride*5 ];

			dest[ word ] = a0;
			if( word + stride < wordCount )
				dest[ word + stride ] = a1;
			if( word + stride*2 < wordCount )
				dest[ word + stride*2 ] = a2;
			if( word + stride*3 < wordCount )
				dest[ word + stride*3 ] = a3;
			if( word + stride*4 < wordCount )
				dest[ word + stride*4 ] = a4;
			if( word + stride*5 < wordCount )
				dest[ word + stride*5 ] = a5;

			word += stride*6;
		}
	}
	else
	{
		/* Input is DWord aligned */

		uint2 *dest = (uint2*) desc.dest;
		uint2 *source = (uint2*) desc.source;
		uint32_t dwordCount = desc.bytes >> 3;
		size_t word = blockIdx.x * blockDim.x + threadIdx.x;
		size_t stride = blockDim.x * totalBlocks;
		while( word < dwordCount )
		{
			uint2 a0, a1, a2, a3, a4, a5;
			a0 = source[ word ];
			if( word + stride < dwordCount )
				a1 = source[ word + stride ];
			if( word + stride*2 < dwordCount )
				a2 = source[ word + stride*2 ];
			if( word + stride*3 < dwordCount )
				a3 = source[ word + stride*3 ];
			if( word + stride*4 < dwordCount )
				a4 = source[ word + stride*4 ];
			if( word + stride*5 < dwordCount )
				a5 = source[ word + stride*5 ];

			dest[ word ] = a0;
			if( word + stride < dwordCount )
				dest[ word + stride ] = a1;
			if( word + stride*2 < dwordCount )
				dest[ word + stride*2 ] = a2;
			if( word + stride*3 < dwordCount )
				dest[ word + stride*3 ] = a3;
			if( word + stride*4 < dwordCount )
				dest[ word + stride*4 ] = a4;
			if( word + stride*5 < dwordCount )
				dest[ word + stride*5 ] = a5;

			word += stride*6;
		}
	}

	__threadfence_system();
}

extern "C" __global__
void MemCopyAsync( physx::PxGpuCopyDesc desc )
{
	performCopy( desc, gridDim.x );
}


extern "C" __global__
void MemCopyBatchedAsync( physx::PxGpuCopyDesc *desc )
{
	__shared__ physx::PxGpuCopyDesc sdesc;

	if( threadIdx.x < sizeof(physx::PxGpuCopyDesc) / sizeof(uint32_t) )
	{
		uint32_t *dest = (uint32_t*)&sdesc;
		uint32_t *source = (uint32_t*)(desc + blockIdx.y);
		dest[ threadIdx.x ] = source[ threadIdx.x ];
		__threadfence_block();
	}
	__syncthreads();

	performCopy( sdesc, gridDim.x );
}
